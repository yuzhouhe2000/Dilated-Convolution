#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "utils_cu.h"

// // Safe free
float* cudaNew_(int size){
	float *ptr =(float*)malloc(sizeof(float)*size);
	float *ptr_cu;
	hipMalloc((void**)&ptr_cu,sizeof(float)*size);
	hipMemcpy(ptr_cu,ptr,sizeof(float)*size,hipMemcpyHostToDevice);
	cudaFree_(ptr);
	return ptr_cu;
}




// Safe free host
void cudaFree_(float* ptr){
	if (ptr) {
		hipFree(ptr);
		ptr = NULL;
	}
}



// cuda data operation
struct tensor_ tensor2gpu(struct tensor_ input){
	float* input_gpu;
	hipMalloc((void **) &input_gpu, sizeof(float)*input.SIZE);
	hipMemcpy(input_gpu,input.data,sizeof(float)*input.SIZE,hipMemcpyHostToDevice);
	free_(input.data);
	input.data = input_gpu;
	return input;
}

struct tensor_ tensor2cpu(struct tensor_ input_gpu){
	float* input = (float*)malloc(sizeof(float) * input_gpu.SIZE);
	hipMemcpy(input,input_gpu.data,sizeof(float)*input_gpu.SIZE,hipMemcpyDeviceToHost);
	cudaFree_(input_gpu.data);
	input_gpu.data = input;
	return input_gpu;
}

struct kernel_ kernel2gpu(struct kernel_ kernel){
	float* kernel_gpu;
	hipMalloc((void **) &kernel_gpu, sizeof(float)*kernel.SIZE);
	hipMemcpy(kernel_gpu,kernel.data,sizeof(float)*kernel.SIZE,hipMemcpyHostToDevice);
	free_(kernel.data);
	kernel.data = kernel_gpu;
	return kernel;
}

struct kernel_ kernel2cpu(struct kernel_ kernel_gpu){
	float* kernel = (float*)malloc(sizeof(float) * kernel_gpu.SIZE);
	hipMemcpy(kernel,kernel_gpu.data,sizeof(float)*kernel_gpu.SIZE,hipMemcpyDeviceToHost);
	cudaFree_(kernel_gpu.data);
	kernel_gpu.data = kernel;
	return kernel_gpu;
}

float* data2gpu(float* input,int SIZE){
	float* input_gpu;
	hipMalloc((void **) &input_gpu, sizeof(float)*SIZE);
	hipMemcpy(input_gpu,input,sizeof(float)*SIZE,hipMemcpyHostToDevice);
	free_(input);
	return input_gpu;
}

float* data2cpu(float* input_gpu,int SIZE){
	float* input = (float*)malloc(sizeof(float) * SIZE);
	hipMemcpy(input,input_gpu,sizeof(float)*SIZE,hipMemcpyDeviceToHost);
	cudaFree_(input_gpu);
	return input;
}


__device__ int find_kernel_idx_gpu(int cout,int cin,int hk, int wk,struct kernel_ kernel){
	return cout * kernel.Cin * kernel.H * kernel.W + cin * kernel.H * kernel.W + hk * kernel.W + wk;
}

__device__ int find_tensor_idx_gpu(int n,int cin, int hin,int win,struct tensor_ input){
	return n * input.H * input.W * input.C + cin*input.W*input.H + hin * input.W  + win;
}

__device__ int find_NCHW_idx_gpu(int n,int cin,int hin,int win,int N,int C,int H,int W){
	return n * H * W * C + cin*H*W +  hin * W + win;
}



// Safe free
__host__ void free_(float* ptr){
	if (ptr) {
		free(ptr);
		ptr = NULL;
	}
}

int check_tensor(struct tensor_ A, struct tensor_ B) {
	if (A.SIZE != B.SIZE) {
		printf("Output SIZE Incorrect!\n");
		return 0;
	}
	for (int i = 0; i < A.SIZE; i++) {
		if (A.data[i] != B.data[i]){
			printf("Output ELEMENT Incorrect!\n");
			return 0;
		}
	}
	printf("Output Correct!\n");
	return 1;
}

float* transpose(float *input, const int N,const int C,const int H, const int W) {
	float* inputT = (float*)malloc(sizeof(float) * N*C*H*W);
	int n;
	#pragma omp parallel for
	for(n = 0; n<N; n++) {
		for(int c = 0; c <C; c++) {
			for(int h = 0; h<H; h++) {
				for (int w = 0; w < W; w++) {
					inputT[n * C * H * W + c * H * W + w*H+h] = input[n * C * H * W + c * H * W + W * h + w];
				}
			}
		}
	}
	return inputT;
}

float* NCHW_2_NHWC(float* input, const int N, const int C, const int H, const int W) {
	float* inputT = (float*)malloc(sizeof(float) * N * C * H * W);
	int n;
	#pragma omp parallel for
	for (n = 0; n < N; n++) {
		for (int c = 0; c < C; c++) {
			for (int h = 0; h < H; h++) {
				for (int w = 0; w < W; w++) {
					inputT[find_NCHW_idx(n,h,w,c,N,H,W,C)] = input[find_NCHW_idx(n, c, h, w, N, C, H, W)];
				}
			}
		}
	}
	return inputT;
}

float* NHWC_2_NCHW(float* input, const int N, const int C, const int H, const int W) {
	float* inputT = (float*)malloc(sizeof(float) * N * C * H * W);
	int n;
	#pragma omp parallel for
	for (n = 0; n < N; n++) {
		for (int h = 0; h < H; h++) {
			for (int w = 0; w < W; w++) {
				for (int c = 0; c < C; c++) {
					inputT[find_NCHW_idx(n, c, h, w, N, C, H, W)] = input[find_NCHW_idx(n, h, w, c, N, H, W, C)];
				}
			}
		}
	}
	return inputT;
}


float* slice(float* input,int start,int end){
	float* input_slice = (float*)malloc(sizeof(float) * (end-start));
	memcpy(input_slice, input + start, (end - start) * sizeof(float));
	return input_slice;
}

int find_kernel_idx(int cout,int cin,int hk, int wk,struct kernel_ kernel){
	return cout * kernel.Cin * kernel.H * kernel.W + cin * kernel.H * kernel.W + hk * kernel.W + wk;
}

int find_tensor_idx(int n,int cin, int hin,int win,struct tensor_ input){
	return n * input.H * input.W * input.C + cin*input.W*input.H + hin * input.W  + win;
}

int find_NCHW_idx(int n,int cin,int hin,int win,int N,int C,int H,int W){
	return n * H * W * C + cin*H*W +  hin * W + win;
}

int find_CCHW_idx(int cout,int cin,int hk,int wk,int Cout,int Cin,int H,int W){
	return cout * Cin * H * W + cin * H * W + hk * W + wk;
}

void print_kernel(struct kernel_ kernel) {
	int kernel_idx;
	for (int cout = 0; cout < kernel.Cout; cout++) {
		for (int cin = 0; cin < kernel.Cin; cin++) {
			for (int hk = 0; hk < kernel.H; hk++) {
				for (int wk = 0; wk < kernel.W; wk++) {
					kernel_idx = find_kernel_idx(cout,cin,hk,wk,kernel);
					printf("[%.f] ", kernel.data[kernel_idx]);
				}
				printf("\n");
			}
			printf("\n");
		}
		printf("\n");
	}

}

void print_CHW(float* input,int C,int H,int W) {
	int kernel_idx;
	for (int cin = 0; cin < C; cin++) {
		for (int hk = 0; hk < H; hk++) {
			for (int wk = 0; wk < W; wk++) {
				kernel_idx = cin*H*W + hk*W + wk;

				printf("[%.2f] ", input[kernel_idx]);
			}
			printf("\n");
		}
		printf("\n");
	}
}

void print_W(float* input,int W) {
	for (int wk = 0; wk < W; wk++) {
		printf("[%.f] ", input[wk]);
	}
	printf("\n");
	printf("\n");
}

void print_tensor(struct tensor_ output) {
	for (int n = 0; n < output.N; n++) {
		for (int cout = 0; cout < output.C; cout++) {
			for (int hout = 0; hout < output.H; hout++) {
				for (int wout = 0; wout < output.W; wout++) {
					printf("[%.f] ", output.data[find_tensor_idx(n,cout,hout,wout,output)]);
				}
				printf("\n");
			}
			printf("\n");
		}
		printf("\n");
	}
	printf("\n");
}

// naive kernel dilation transformation on CPU
__host__ struct kernel_ kernel_simple_dilation(struct kernel_ kernel){
	if ((kernel.dilH == 1)&&(kernel.dilW == 1)){
		return kernel;
	}
	int posH, posW, kernel_idx, kernel_idx_new;
	// // dilation method 1: 0101010    dil(3,2) = 7
	// int newH = (kernel.dilH - 1) * (kernel.H + 1) + kernel.H;
	// int newW = (kernel.dilW - 1) * (kernel.W + 1) + kernel.W;
	// dilation method 2: 10101		dil(3,2) = 5

	int newH = (kernel.dilH-1) * (kernel.H-1)+kernel.H;
	int newW = (kernel.dilW-1) * (kernel.W-1)+kernel.W;
	int newSize = kernel.Cout * newH * newW * kernel.Cin;
	float* B_dil = (float*)malloc(sizeof(float) * newSize);
	for (int i = 0; i < newSize; i++) {
		B_dil[i] = 0.0f;
	}
	for (int cout = 0; cout < kernel.Cout;cout++){
		for (int cin = 0; cin < kernel.Cin;cin++){
			for (int hk = 0; hk < kernel.H;hk++){
				for (int wk = 0; wk < kernel.W;wk++){
					// // dilation 1
					// posH = ((hk+1)*kernel.dilH-1);
					// posW = ((wk+1)*kernel.dilW-1);
					// dilation 2
					posH = hk*kernel.dilH;
					posW = wk*kernel.dilW;
					kernel_idx = find_kernel_idx(cout,cin,hk,wk,kernel);
					kernel_idx_new = find_CCHW_idx(cout,cin,posH,posW,kernel.Cout,kernel.Cin,newH,newW);
					B_dil[kernel_idx_new] = kernel.data[kernel_idx];
				}
			}
		}
	}
	struct kernel_ kernel_new = { .data = B_dil, .Cout = kernel.Cout, .Cin = kernel.Cin, .H = newH, .W = newW, .dilH = 1, .dilW = 1, .padH = kernel.padH, .padW = kernel.padW, .strideH = kernel.strideH, .strideW = kernel.strideW,.SIZE = newSize};
	//print_kernel(kernel_new);
	return kernel_new;
} 



// pad tensor
struct tensor_ tensor_pad(struct tensor_ input,int padH,int padW){
	if ((padH == 0)&&(padW == 0)){
		return input;
	}
	int kernel_idx;
	int newH = input.H+2*padH;
	int newW = input.W+2*padW;
	int newSize = input.C * newH * newW * input.N;
	float* output = (float*)malloc(sizeof(float) * newSize);
	for (int n = 0; n < input.N; n++) {
		for (int hout = 0; hout < newH; hout++) {
			for (int wout = 0; wout < newW; wout++) {
				for (int cout = 0; cout < input.C; cout++) {
					if (hout < padH || hout >= (newH-padH)|| wout < padW || wout >= (newW - padW)){
						output[find_NCHW_idx(n,cout,hout,wout,input.N,input.C,newH,newW)] = 0;
					}
					else{
						kernel_idx = find_tensor_idx(n,cout,hout-padH,wout-padW,input);
						output[find_NCHW_idx(n, cout, hout, wout, input.N, input.C, newH, newW)] = input.data[kernel_idx];
					}
				}
			}
		}
	}
	struct tensor_ padded = { .data = output,.N = input.N, .H = newH, .W = newW,  .C = input.C,.SIZE = newH*newW*input.N*input.C};
	print_tensor(padded);
	return padded;
} 

