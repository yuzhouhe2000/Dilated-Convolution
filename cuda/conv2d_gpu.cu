#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "utils_cu.h"
#include <string.h>
// #include <winograd_transform.h>


// Golden Output. Naive implementation of direct convolution on CPU. Tensor in NCHW layout.
struct tensor_ conv2d_direct_convolution_cpu(struct tensor_ input, struct kernel_ kernel_raw){
	struct kernel_ kernel = kernel_simple_dilation(kernel_raw);
	int Hout = ((input.H + 2*kernel.padH - kernel.dilH * (kernel.H - 1) - 1)/kernel.strideH) + 1;
	int Wout = ((input.W + 2*kernel.padW - kernel.dilW * (kernel.W - 1) - 1)/kernel.strideW) + 1;
	float* C = (float*)malloc(sizeof(float) * input.N * Hout * Wout * kernel.Cout);
	int n;
	for (n = 0; n < input.N;n++){
		for (int cout = 0; cout < kernel.Cout; cout++){
			for (int hout = 0; hout < Hout; hout++){
				for (int wout = 0; wout < Wout; wout++){
					float accum = 0.0f;
					for (int cin = 0; cin < input.C;cin++){
						for (int hk = 0; hk < kernel.H;hk=hk+2){
							for (int wk = 0; wk < kernel.W;wk=wk+2){	
								int hin = (hout * kernel.strideH + hk)-kernel.padH;
								int win = (wout * kernel.strideW + wk)-kernel.padW;
								if (hin < 0 || hin >= input.H || win < 0 || win >= input.W) {
									accum += 0;
								}
								else {
									int input_idx = find_tensor_idx(n, cin, hin, win, input);
									int kernel_idx = find_kernel_idx(cout, cin, hk, wk, kernel);
									accum += (input.data[input_idx] * kernel.data[kernel_idx]);	
								}
							}
						}
					}
					C[find_NCHW_idx(n,cout,hout,wout,input.N,kernel.Cout,Hout,Wout)] = accum;
				}
			}
		}
	}
	free(kernel.data);
	struct tensor_ output = { .data = C,.N = input.N, .H = Hout, .W = Wout,  .C = kernel.Cout,.SIZE = Hout*Wout*input.N*kernel.Cout};
	return output;
}

// Naive implementation of direct convolution on CPU. Tensor in NCHW layout.
__global__ void conv2d_direct_convolution_gpu(struct tensor_ input, struct kernel_ kernel, struct tensor_ output){
	// Distribute computation of each output pixel to a separate cuda kernel
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int n = idx/(kernel.Cout * output.H * output.W);
	int cout = ((idx% (kernel.Cout * output.H * output.W))/(output.H * output.W)) ;
	int hout = ((idx% (output.H*output.W))/(output.W)) ;
	int wout = idx%(output.W);
	float accum = 0.0f;
	for (int cin = 0; cin < input.C;cin++){
		for (int hk = 0; hk < kernel.H;hk=hk+2){
			for (int wk = 0; wk < kernel.W;wk=wk+2){	
				int hin = (hout * kernel.strideH + hk)-kernel.padH;
				int win = (wout * kernel.strideW + wk)-kernel.padW;
				int input_idx = n * input.H * input.W * input.C + cin*input.W*input.H + hin * input.W  + win;
				int kernel_idx = cout * kernel.Cin * kernel.H * kernel.W + cin * kernel.H * kernel.W + hk * kernel.W + wk;
				accum += (input.data[input_idx] * kernel.data[kernel_idx]);	
				
			}
		}
	}
	output.data[n * kernel.Cout * output.H * output.W + cout*output.H*output.W +  hout * output.W + wout] = accum;
}



struct tensor_ conv2d_dilated_winograd23s1d2_cpu1(struct tensor_ input_raw, struct kernel_ kernel){
	// TODO: use better way to pad
	struct tensor_ input = tensor_pad(input_raw, kernel.padH, kernel.padW);
	kernel.padH = 0;
	kernel.padW = 0;
	int Hout = ((input.H + 2*kernel.padH - kernel.dilH * (kernel.H - 1) - 1)/kernel.strideH) + 1;
	int Wout = ((input.W + 2*kernel.padW - kernel.dilW * (kernel.W - 1) - 1)/kernel.strideW) + 1;
	float* C = (float*)malloc(sizeof(float) * input.N * Hout * Wout * kernel.Cout);
	// 3x3 kernel dilation -> 5x5
	struct kernel_ dilated_kernel = kernel_simple_dilation(kernel);
	// For each batch
	int n;
	#pragma omp parallel for
	for (n = 0; n < input.N;n++){
		float* A_n = slice(input.data,n* input.H * input.W * input.C, (n+1) * input.H * input.W * input.C);
		// For each tile group (4 tiles)
		// Overlap = 4		
		for (int hin = 0; hin < input.H-7; hin=hin+4){
			for (int win = 0; win < input.W-7; win=win+4){
				// Initialize tile group (4 tiles)
				// TODO: maybe NHWC better? Transpose?
				float* tile_group = (float*)malloc(sizeof(float) * input.C * 8*8);
				for (int cin = 0; cin < input.C; cin++){		
					for (int yy = 0; yy < 8; yy++){
						for (int xx = 0; xx < 8; xx++){
							int posH = hin+yy;
							int posW = win+xx;
							int A_n_idx = cin*input.H*input.W+posH*input.W+posW;
							tile_group[cin*64+yy*8+xx] = A_n[A_n_idx];
						}
					}
				}
				//print_CHW(tile_group, kernel.Cin, 8, 8);
				// winograd on tile
				// output matrix size = 4x4, input matrix size = 8x8, kernel size = 5x5
				float* tile_output = tile_wino23s1d2_cpu(tile_group,dilated_kernel,Hout,Wout);
				free_(tile_group);
				// memcpy tile result to output matrix C
				for (int cout = 0; cout < kernel.Cout; cout++){		
					for (int yy = 0; yy < 4; yy++){
						for (int xx = 0; xx < 4; xx++){
							int posH = hin+yy;
							int posW = win+xx;
							int tile_idx = cout * 16 + yy * 4 + xx;
							int C_idx = find_NCHW_idx(n, cout, posH, posW, input.N, kernel.Cout, Hout, Wout);
							C[C_idx] = tile_output[tile_idx];
						}
					}
				}
				free_(tile_output);
			}
		}	
		free_(A_n);
	}	
	struct tensor_ output = { .data = C, .H = Hout, .W = Wout, .N = input.N, .C = kernel.Cout,.SIZE = Hout*Wout*input.N*kernel.Cout};
	return output;
}



int main(){		
	int N = 4;
	int Hin = 128;
	int Win = 128;
	int Cin = 16;
	int Cout = 16;
	int Hk = 3;
	int Wk = 3;
	int dilH = 2;
	int dilW = 2;
	int padH = 0;
	int padW = 0;
	int strideH = 1;
	int strideW = 1;

	printf("N = %d\n",N);
	printf("Cout = %d\n",Cout);
	printf("Cin = %d\n",Cin);
	printf("Hin = %d\n",Hin);
	printf("Win = %d\n",Win);

	float* A = (float*)malloc(sizeof(float) * N * Hin * Win * Cin);
	float* B = (float*)malloc(sizeof(float) * Cout * Hk * Wk * Cin);
	struct tensor_ input = { .data = A, .N = N, .H = Hin, .W = Win,  .C = Cin,.SIZE = Hin * Win * N * Cin };
	struct kernel_ kernel = { .data = B, .Cout = Cout, .Cin = Cin, .H = Hk, .W = Wk, .dilH = dilH, .dilW = dilW, .padH = padH, .padW = padW, .strideH = strideH, .strideW = strideW,.SIZE = Cout * Hk * Wk* Cin};

	for (int i = 0; i <input.SIZE; i++) {
		input.data[i] = 2;
	}
	for (int i = 0; i < kernel.SIZE; i++) {
		kernel.data[i] = 2;
	}
	struct tensor_ output2 = conv2d_direct_convolution_cpu(input,kernel);
	// print_tensor(output2);
	struct kernel_ kernel_ = kernel_simple_dilation(kernel);
	
	// output
	int Hout = ((input.H + 2*kernel.padH - kernel.dilH * (kernel.H - 1) - 1)/kernel.strideH) + 1;
	int Wout = ((input.W + 2*kernel.padW - kernel.dilW * (kernel.W - 1) - 1)/kernel.strideW) + 1;

	// load data to gpu
	float* output_data = (float*)malloc(sizeof(float) * input.N * Hout * Wout * kernel.Cout);
	struct tensor_ output = { .data = output_data, .N = input.N,.H = Hout, .W = Wout,  .C = kernel.Cout,.SIZE = Hout*Wout*input.N*kernel.Cout};
	struct kernel_ kernel_gpu = kernel2gpu(kernel_);
	struct tensor_ input_gpu = tensor2gpu(input);
	struct tensor_ output_gpu = tensor2gpu(output);
	struct timespec start, stop;
	double time;

	if(clock_gettime(CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}	

	int THREAD = N*kernel.Cout*Hout*Wout;
	dim3 dimGrid(961);
	dim3 dimBlock(1024);
	conv2d_direct_convolution_gpu<<< dimGrid, dimBlock>>> (input_gpu,kernel_gpu,output_gpu);

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	 
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);

	output = tensor2cpu(output_gpu);
	// print_tensor(output);	
	check_tensor(output, output2);





	free_(kernel.data);
	// free(input.data);
	cudaFree_(input_gpu.data);
	cudaFree_(kernel_.data);
	free_(output.data);
	
	return 0;
}	




