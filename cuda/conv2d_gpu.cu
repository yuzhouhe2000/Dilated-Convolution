#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "utils_cu.h"
#include <string.h>
// #include <winograd_transform.h>


// Golden Output. Naive implementation of direct convolution on CPU. Tensor in NCHW layout.
struct tensor_ conv2d_direct_convolution_cpu(struct tensor_ input, struct kernel_ kernel_raw){
	struct kernel_ kernel = kernel_simple_dilation(kernel_raw);
	int Hout = ((input.H + 2*kernel.padH - kernel.dilH * (kernel.H - 1) - 1)/kernel.strideH) + 1;
	int Wout = ((input.W + 2*kernel.padW - kernel.dilW * (kernel.W - 1) - 1)/kernel.strideW) + 1;
	float* C = (float*)malloc(sizeof(float) * input.N * Hout * Wout * kernel.Cout);
	int n;
	for (n = 0; n < input.N;n++){
		for (int cout = 0; cout < kernel.Cout; cout++){
			for (int hout = 0; hout < Hout; hout++){
				for (int wout = 0; wout < Wout; wout++){
					float accum = 0.0f;
					for (int cin = 0; cin < input.C;cin++){
						for (int hk = 0; hk < kernel.H;hk=hk+2){
							for (int wk = 0; wk < kernel.W;wk=wk+2){	
								int hin = (hout * kernel.strideH + hk)-kernel.padH;
								int win = (wout * kernel.strideW + wk)-kernel.padW;
								if (hin < 0 || hin >= input.H || win < 0 || win >= input.W) {
									accum += 0;
								}
								else {
									int input_idx = find_tensor_idx(n, cin, hin, win, input);
									int kernel_idx = find_kernel_idx(cout, cin, hk, wk, kernel);
									accum += (input.data[input_idx] * kernel.data[kernel_idx]);	
								}
							}
						}
					}
					C[find_NCHW_idx(n,cout,hout,wout,input.N,kernel.Cout,Hout,Wout)] = accum;
				}
			}
		}
	}
	free(kernel.data);
	struct tensor_ output = { .data = C,.N = input.N, .H = Hout, .W = Wout,  .C = kernel.Cout,.SIZE = Hout*Wout*input.N*kernel.Cout};
	return output;
}

// Naive implementation of direct convolution on CPU. Tensor in NCHW layout.
__global__ void conv2d_direct_convolution_gpu(struct tensor_ input, struct kernel_ kernel, struct tensor_ output){
	// Distribute computation of each output pixel to a separate cuda kernel
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int n = idx/(kernel.Cout * output.H * output.W);
	int cout = ((idx% (kernel.Cout * output.H * output.W))/(output.H * output.W)) ;
	int hout = ((idx% (output.H*output.W))/(output.W)) ;
	int wout = idx%(output.W);
	float accum = 0.0f;
	for (int cin = 0; cin < input.C;cin++){
		for (int hk = 0; hk < kernel.H;hk=hk+2){
			for (int wk = 0; wk < kernel.W;wk=wk+2){	
				int hin = (hout * kernel.strideH + hk)-kernel.padH;
				int win = (wout * kernel.strideW + wk)-kernel.padW;
				int input_idx = n * input.H * input.W * input.C + cin*input.W*input.H + hin * input.W  + win;
				int kernel_idx = cout * kernel.Cin * kernel.H * kernel.W + cin * kernel.H * kernel.W + hk * kernel.W + wk;
				accum += (input.data[input_idx] * kernel.data[kernel_idx]);	
				
			}
		}
	}
	output.data[n * kernel.Cout * output.H * output.W + cout*output.H*output.W +  hout * output.W + wout] = accum;
}


int main(){		
	int N = 4;
	int Hin = 128;
	int Win = 128;
	int Cin = 16;
	int Cout = 16;
	int Hk = 3;
	int Wk = 3;
	int dilH = 2;
	int dilW = 2;
	int padH = 0;
	int padW = 0;
	int strideH = 1;
	int strideW = 1;

	printf("N = %d\n",N);
	printf("Cout = %d\n",Cout);
	printf("Cin = %d\n",Cin);
	printf("Hin = %d\n",Hin);
	printf("Win = %d\n",Win);

	float* A = (float*)malloc(sizeof(float) * N * Hin * Win * Cin);
	float* B = (float*)malloc(sizeof(float) * Cout * Hk * Wk * Cin);
	struct tensor_ input = { .data = A, .N = N, .H = Hin, .W = Win,  .C = Cin,.SIZE = Hin * Win * N * Cin };
	struct kernel_ kernel = { .data = B, .Cout = Cout, .Cin = Cin, .H = Hk, .W = Wk, .dilH = dilH, .dilW = dilW, .padH = padH, .padW = padW, .strideH = strideH, .strideW = strideW,.SIZE = Cout * Hk * Wk* Cin};

	for (int i = 0; i <input.SIZE; i++) {
		input.data[i] = 2;
	}
	for (int i = 0; i < kernel.SIZE; i++) {
		kernel.data[i] = 2;
	}
	struct tensor_ output2 = conv2d_direct_convolution_cpu(input,kernel);
	// print_tensor(output2);
	struct kernel_ kernel_ = kernel_simple_dilation(kernel);
	
	// output
	int Hout = ((input.H + 2*kernel.padH - kernel.dilH * (kernel.H - 1) - 1)/kernel.strideH) + 1;
	int Wout = ((input.W + 2*kernel.padW - kernel.dilW * (kernel.W - 1) - 1)/kernel.strideW) + 1;

	// load data to gpu
	float* output_data = (float*)malloc(sizeof(float) * input.N * Hout * Wout * kernel.Cout);
	struct tensor_ output = { .data = output_data, .N = input.N,.H = Hout, .W = Wout,  .C = kernel.Cout,.SIZE = Hout*Wout*input.N*kernel.Cout};
	struct kernel_ kernel_gpu = kernel2gpu(kernel_);
	struct tensor_ input_gpu = tensor2gpu(input);
	struct tensor_ output_gpu = tensor2gpu(output);
	struct timespec start, stop;
	double time;

	if(clock_gettime(CLOCK_REALTIME, &start) == -1 ) { perror( "clock gettime" );}	

	int THREAD = N*kernel.Cout*Hout*Wout;
	dim3 dimGrid(961);
	dim3 dimBlock(1024);
	conv2d_direct_convolution_gpu<<< dimGrid, dimBlock>>> (input_gpu,kernel_gpu,output_gpu);

	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) { perror( "clock gettime" );}	 
	time = (stop.tv_sec - start.tv_sec)+ (double)(stop.tv_nsec - start.tv_nsec)/1e9;
	printf("time is %f ns\n", time*1e9);

	output = tensor2cpu(output_gpu);
	// print_tensor(output);	
	check_tensor(output, output2);





	free_(kernel.data);
	// free(input.data);
	cudaFree_(input_gpu.data);
	cudaFree_(kernel_.data);
	free_(output.data);
	
	return 0;
}	




